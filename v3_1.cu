#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.01
#define EPOCHS 3
#define BATCH_SIZE 64
#define NUM_CLASSES 10

// Check CUDA errors
#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Timer function for CPU
double get_cpu_time(clock_t start) {
    return (double)(clock() - start) / CLOCKS_PER_SEC;
}

// Timer function for GPU
float get_gpu_time(hipEvent_t start, hipEvent_t stop) {
    float milliseconds = 0;
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    return milliseconds / 1000.0; // Convert to seconds
}

// Allocate memory for a matrix on host
double** allocateMatrix(int rows, int cols) {
    double** mat = (double**)malloc(rows * sizeof(double*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (double*)malloc(cols * sizeof(double));
    }
    return mat;
}

// Free allocated matrix memory on host
void freeMatrix(double** mat, int rows) {
    for (int i = 0; i < rows; i++) {
        free(mat[i]);
    }
    free(mat);
}

// Activation functions (CPU)
void relu_cpu(double* x, int size) {
    for (int i = 0; i < size; i++) {
        x[i] = (x[i] > 0) ? x[i] : 0;
    }
}

void softmax_cpu(double* x, int size) {
    double max = x[0];
    for (int i = 1; i < size; i++) {
        if (x[i] > max) max = x[i];
    }
    double sum = 0;
    for (int i = 0; i < size; i++) {
        x[i] = exp(x[i] - max);
        sum += x[i];
    }
    for (int i = 0; i < size; i++) {
        x[i] /= sum;
    }
}

// CUDA kernels
__global__ void matrixMulKernel(double* A, double* B, double* C, double* bias, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rowsA && col < colsB) {
        double sum = bias[row];
        for (int k = 0; k < colsA; k++) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

__global__ void reluKernel(double* x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] = (x[idx] > 0) ? x[idx] : 0;
    }
}

__global__ void softmaxKernel(double* x, int size) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        double max = x[0];
        for (int i = 1; i < size; i++) {
            if (x[i] > max) max = x[i];
        }
        double sum = 0;
        for (int i = 0; i < size; i++) {
            x[i] = exp(x[i] - max);
            sum += x[i];
        }
        for (int i = 0; i < size; i++) {
            x[i] /= sum;
        }
    }
}

__global__ void outputGradientKernel(double* output, double* target, double* d_output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_output[idx] = output[idx] - target[idx];
    }
}

__global__ void hiddenGradientKernel(double* W2, double* d_output, double* hidden, double* d_hidden, int hidden_size, int output_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < hidden_size) {
        double sum = 0;
        for (int j = 0; j < output_size; j++) {
            sum += W2[j * hidden_size + i] * d_output[j];
        }
        d_hidden[i] = sum * (hidden[i] > 0);
    }
}

__global__ void updateWeightsKernel(double* W, double* grad, double* input, int rows, int cols, double lr) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows && j < cols) {
        W[i * cols + j] -= lr * grad[i] * input[j];
    }
}

__global__ void updateBiasKernel(double* b, double* grad, int size, double lr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        b[i] -= lr * grad[i];
    }
}

// Neural network structure
typedef struct {
    double** W1;
    double** W2;
    double* b1;
    double* b2;
    double *d_W1, *d_W2, *d_b1, *d_b2;
} NeuralNetwork;

// Initialize neural network
NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    net->W1 = allocateMatrix(HIDDEN_SIZE, INPUT_SIZE);
    net->W2 = allocateMatrix(OUTPUT_SIZE, HIDDEN_SIZE);
    net->b1 = (double*)calloc(HIDDEN_SIZE, sizeof(double));
    net->b2 = (double*)calloc(OUTPUT_SIZE, sizeof(double));

    srand(42);
    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1[i][j] = ((double)rand() / RAND_MAX) * 0.01;

    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2[i][j] = ((double)rand() / RAND_MAX) * 0.01;

    CUDA_CHECK(hipMalloc(&net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->d_b1, HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->d_b2, OUTPUT_SIZE * sizeof(double)));

    double* temp_W1;
    double* temp_W2;
    CUDA_CHECK(hipHostMalloc(&temp_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&temp_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));
    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            temp_W1[i * INPUT_SIZE + j] = net->W1[i][j];
    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            temp_W2[i * HIDDEN_SIZE + j] = net->W2[i][j];

    CUDA_CHECK(hipMemcpy(net->d_W1, temp_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(net->d_W2, temp_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(net->d_b1, net->b1, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(net->d_b2, net->b2, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

    CUDA_CHECK(hipHostFree(temp_W1));
    CUDA_CHECK(hipHostFree(temp_W2));

    return net;
}

// Forward pass (CPU)
void forward_cpu(NeuralNetwork* net, double* input, double* hidden, double* output) {
    for (int i = 0; i < HIDDEN_SIZE; i++) {
        hidden[i] = net->b1[i];
        for (int j = 0; j < INPUT_SIZE; j++)
            hidden[i] += net->W1[i][j] * input[j];
    }
    relu_cpu(hidden, HIDDEN_SIZE);

    for (int i = 0; i < OUTPUT_SIZE; i++) {
        output[i] = net->b2[i];
        for (int j = 0; j < HIDDEN_SIZE; j++)
            output[i] += net->W2[i][j] * hidden[j];
    }
    softmax_cpu(output, OUTPUT_SIZE);
}

// Forward pass (GPU)
void forward_gpu(NeuralNetwork* net, double* d_input, double* d_hidden, double* d_output) {
    dim3 blockDim(16, 16);
    dim3 gridDim((1 + 15) / 16, (HIDDEN_SIZE + 15) / 16);
    matrixMulKernel<<<gridDim, blockDim>>>(net->d_W1, d_input, d_hidden, net->d_b1, HIDDEN_SIZE, INPUT_SIZE, 1);
    CUDA_CHECK(hipDeviceSynchronize());

    int threads = 256;
    int blocks = (HIDDEN_SIZE + threads - 1) / threads;
    reluKernel<<<blocks, threads>>>(d_hidden, HIDDEN_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    gridDim = dim3((1 + 15) / 16, (OUTPUT_SIZE + 15) / 16);
    matrixMulKernel<<<gridDim, blockDim>>>(net->d_W2, d_hidden, d_output, net->d_b2, OUTPUT_SIZE, HIDDEN_SIZE, 1);
    CUDA_CHECK(hipDeviceSynchronize());

    softmaxKernel<<<1, 32>>>(d_output, OUTPUT_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());
}

// Backward pass (CPU)
void backward_cpu(NeuralNetwork* net, double* input, double* hidden, double* output, double* target) {
    double d_output[OUTPUT_SIZE], d_hidden[HIDDEN_SIZE];

    for (int i = 0; i < OUTPUT_SIZE; i++)
        d_output[i] = output[i] - target[i];

    for (int i = 0; i < HIDDEN_SIZE; i++) {
        d_hidden[i] = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++)
            d_hidden[i] += net->W2[j][i] * d_output[j];
        d_hidden[i] *= (hidden[i] > 0);
    }

    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2[i][j] -= LEARNING_RATE * d_output[i] * hidden[j];

    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1[i][j] -= LEARNING_RATE * d_hidden[i] * input[j];

    for (int i = 0; i < OUTPUT_SIZE; i++)
        net->b2[i] -= LEARNING_RATE * d_output[i];

    for (int i = 0; i < HIDDEN_SIZE; i++)
        net->b1[i] -= LEARNING_RATE * d_hidden[i];
}

// Backward pass (GPU)
void backward_gpu(NeuralNetwork* net, double* d_input, double* d_hidden, double* d_output, double* d_target, double* d_d_output, double* d_d_hidden) {
    int threads = 256;
    int blocks = (OUTPUT_SIZE + threads - 1) / threads;
    outputGradientKernel<<<blocks, threads>>>(d_output, d_target, d_d_output, OUTPUT_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    blocks = (HIDDEN_SIZE + threads - 1) / threads;
    hiddenGradientKernel<<<blocks, threads>>>(net->d_W2, d_d_output, d_hidden, d_d_hidden, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    dim3 blockDim(16, 16);
    dim3 gridDim((HIDDEN_SIZE + 15) / 16, (OUTPUT_SIZE + 15) / 16);
    updateWeightsKernel<<<gridDim, blockDim>>>(net->d_W2, d_d_output, d_hidden, OUTPUT_SIZE, HIDDEN_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());

    gridDim = dim3((INPUT_SIZE + 15) / 16, (HIDDEN_SIZE + 15) / 16);
    updateWeightsKernel<<<gridDim, blockDim>>>(net->d_W1, d_d_hidden, d_input, HIDDEN_SIZE, INPUT_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());

    blocks = (OUTPUT_SIZE + threads - 1) / threads;
    updateBiasKernel<<<blocks, threads>>>(net->d_b2, d_d_output, OUTPUT_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());

    blocks = (HIDDEN_SIZE + threads - 1) / threads;
    updateBiasKernel<<<blocks, threads>>>(net->d_b1, d_d_hidden, HIDDEN_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());
}

// Train network
void train(NeuralNetwork* net, double** images, double** labels, int numImages, bool use_gpu, double* total_time, double* loss_out, double* train_acc_out) {
    double loss = 0.0;
    int correct = 0;

    if (use_gpu) {
        // Allocate GPU memory for the entire dataset
        double *d_images, *d_labels;
        CUDA_CHECK(hipMalloc(&d_images, numImages * INPUT_SIZE * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_labels, numImages * OUTPUT_SIZE * sizeof(double)));
        double* temp_W1;
        double* temp_W2;
        CUDA_CHECK(hipHostMalloc(&temp_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));
        CUDA_CHECK(hipHostMalloc(&temp_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));

        // Allocate reusable GPU memory for intermediates
        double *d_hidden, *d_output, *d_d_output, *d_d_hidden;
        CUDA_CHECK(hipMalloc(&d_hidden, HIDDEN_SIZE * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_output, OUTPUT_SIZE * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_d_output, OUTPUT_SIZE * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_d_hidden, HIDDEN_SIZE * sizeof(double)));

        // Copy images and labels to GPU once
        double* temp_images;
        double* temp_labels;
        CUDA_CHECK(hipHostMalloc(&temp_images, numImages * INPUT_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));
        CUDA_CHECK(hipHostMalloc(&temp_labels, numImages * OUTPUT_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));
        for (int i = 0; i < numImages; i++) {
            for (int j = 0; j < INPUT_SIZE; j++) {
                temp_images[i * INPUT_SIZE + j] = images[i][j];
            }
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                temp_labels[i * OUTPUT_SIZE + j] = labels[i][j];
            }
        }

        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        CUDA_CHECK(hipEventRecord(start));

        CUDA_CHECK(hipMemcpy(d_images, temp_images, numImages * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_labels, temp_labels, numImages * OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

        for (int epoch = 0; epoch < EPOCHS; epoch++) {
            loss = 0.0;
            correct = 0;

            for (int i = 0; i < numImages; i++) {
                double output_host[OUTPUT_SIZE]; // For loss and accuracy computation
                double* d_input = d_images + i * INPUT_SIZE;
                double* d_target = d_labels + i * OUTPUT_SIZE;

                forward_gpu(net, d_input, d_hidden, d_output);
                backward_gpu(net, d_input, d_hidden, d_output, d_target, d_d_output, d_d_hidden);

                // Copy output to host for loss and accuracy
                CUDA_CHECK(hipMemcpy(output_host, d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

                for (int k = 0; k < OUTPUT_SIZE; k++) {
                    loss -= labels[i][k] * log(output_host[k] + 1e-10);
                }
                int pred = 0, actual = 0;
                for (int j = 0; j < OUTPUT_SIZE; j++) {
                    if (output_host[j] > output_host[pred]) pred = j;
                    if (labels[i][j] > labels[i][actual]) actual = j;
                }
                if (pred == actual) correct++;
            }

            printf("GPU Epoch %d - Loss: %.4f - Train Accuracy: %.2f%%\n",
                   epoch + 1, loss / numImages, (correct / (double)numImages) * 100);
            if (epoch == EPOCHS - 1) {
                *loss_out = loss / numImages;
                *train_acc_out = (correct / (double)numImages) * 100;
            }
        }

        // Copy weights and biases back to host once
        CUDA_CHECK(hipMemcpy(temp_W1, net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(temp_W2, net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(net->b1, net->d_b1, HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(net->b2, net->d_b2, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

        CUDA_CHECK(hipEventRecord(stop));
        *total_time = get_gpu_time(start, stop);
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));

        for (int i = 0; i < HIDDEN_SIZE; i++)
            for (int j = 0; j < INPUT_SIZE; j++)
                net->W1[i][j] = temp_W1[i * INPUT_SIZE + j];
        for (int i = 0; i < OUTPUT_SIZE; i++)
            for (int j = 0; j < HIDDEN_SIZE; j++)
                net->W2[i][j] = temp_W2[i * HIDDEN_SIZE + j];

        CUDA_CHECK(hipHostFree(temp_W1));
        CUDA_CHECK(hipHostFree(temp_W2));
        CUDA_CHECK(hipHostFree(temp_images));
        CUDA_CHECK(hipHostFree(temp_labels));
        // Free GPU memory
        CUDA_CHECK(hipFree(d_images));
        CUDA_CHECK(hipFree(d_labels));
        CUDA_CHECK(hipFree(d_hidden));
        CUDA_CHECK(hipFree(d_output));
        CUDA_CHECK(hipFree(d_d_output));
        CUDA_CHECK(hipFree(d_d_hidden));
    } else {
        clock_t total_start = clock();

        for (int epoch = 0; epoch < EPOCHS; epoch++) {
            loss = 0.0;
            correct = 0;

            for (int i = 0; i < numImages; i++) {
                double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
                forward_cpu(net, images[i], hidden, output);
                backward_cpu(net, images[i], hidden, output, labels[i]);

                for (int k = 0; k < OUTPUT_SIZE; k++) {
                    loss -= labels[i][k] * log(output[k] + 1e-10);
                }
                int pred = 0, actual = 0;
                for (int j = 0; j < OUTPUT_SIZE; j++) {
                    if (output[j] > output[pred]) pred = j;
                    if (labels[i][j] > labels[i][actual]) actual = j;
                }
                if (pred == actual) correct++;
            }

            printf("CPU Epoch %d - Loss: %.4f - Train Accuracy: %.2f%%\n",
                   epoch + 1, loss / numImages, (correct / (double)numImages) * 100);
            if (epoch == EPOCHS - 1) {
                *loss_out = loss / numImages;
                *train_acc_out = (correct / (double)numImages) * 100;
            }
        }

        *total_time = get_cpu_time(total_start);
        printf("CPU Total training time: %.3fs\n", *total_time);
    }
}

// Evaluate accuracy on test data
void evaluate(NeuralNetwork* net, double** images, double** labels, int numImages, bool use_gpu, double* test_acc_out) {
    int correct = 0;
    if (use_gpu) {
        // Allocate GPU memory for test dataset
        double *d_images, *d_labels;
        CUDA_CHECK(hipMalloc(&d_images, numImages * INPUT_SIZE * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_labels, numImages * OUTPUT_SIZE * sizeof(double)));

        // Allocate reusable GPU memory for intermediates
        double *d_hidden, *d_output;
        CUDA_CHECK(hipMalloc(&d_hidden, HIDDEN_SIZE * sizeof(double)));
        CUDA_CHECK(hipMalloc(&d_output, OUTPUT_SIZE * sizeof(double)));

        // Copy test images and labels to GPU
        double* temp_images;
        double* temp_labels;
        CUDA_CHECK(hipHostMalloc(&temp_images, numImages * INPUT_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));
        CUDA_CHECK(hipHostMalloc(&temp_labels, numImages * OUTPUT_SIZE * sizeof(double), hipHostMallocDefault, hipHostMallocDefault));
        for (int i = 0; i < numImages; i++) {
            for (int j = 0; j < INPUT_SIZE; j++) {
                temp_images[i * INPUT_SIZE + j] = images[i][j];
            }
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                temp_labels[i * OUTPUT_SIZE + j] = labels[i][j];
            }
        }
        CUDA_CHECK(hipMemcpy(d_images, temp_images, numImages * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_labels, temp_labels, numImages * OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipHostFree(temp_images));
        CUDA_CHECK(hipHostFree(temp_labels));

        for (int i = 0; i < numImages; i++) {
            double output_host[OUTPUT_SIZE];
            double* d_input = d_images + i * INPUT_SIZE;
            forward_gpu(net, d_input, d_hidden, d_output);
            CUDA_CHECK(hipMemcpy(output_host, d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

            int pred = 0, actual = 0;
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (output_host[j] > output_host[pred]) pred = j;
                if (labels[i][j] > labels[i][actual]) actual = j;
            }
            if (pred == actual) correct++;
        }

        CUDA_CHECK(hipFree(d_images));
        CUDA_CHECK(hipFree(d_labels));
        CUDA_CHECK(hipFree(d_hidden));
        CUDA_CHECK(hipFree(d_output));
    } else {
        for (int i = 0; i < numImages; i++) {
            double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
            forward_cpu(net, images[i], hidden, output);
            int pred = 0, actual = 0;
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (output[j] > output[pred]) pred = j;
                if (labels[i][j] > labels[i][actual]) actual = j;
            }
            if (pred == actual) correct++;
        }
    }
    *test_acc_out = (correct / (double)numImages) * 100;
    printf("%s Test Accuracy: %.2f%%\n", use_gpu ? "GPU" : "CPU", *test_acc_out);
}

// Read MNIST dataset
double** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 16, SEEK_SET);
    double** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;
            if (fread(&pixel, sizeof(unsigned char), 1, file) != 1) {
                fprintf(stderr, "Error: Failed to read pixel\n");
                fclose(file);
                exit(EXIT_FAILURE);
            }
            images[i][j] = pixel / 255.0;
        }
    }
    fclose(file);
    return images;
}

double** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 8, SEEK_SET);
    double** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        if (fread(&label, sizeof(unsigned char), 1, file) != 1) {
            fprintf(stderr, "Error: Failed to read label\n");
            fclose(file);
            exit(EXIT_FAILURE);
        }
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0 : 0.0;
        }
    }
    fclose(file);
    return labels;
}

// Free network memory
void freeNetwork(NeuralNetwork* net) {
    freeMatrix(net->W1, HIDDEN_SIZE);
    freeMatrix(net->W2, OUTPUT_SIZE);
    free(net->b1);
    free(net->b2);
    CUDA_CHECK(hipFree(net->d_W1));
    CUDA_CHECK(hipFree(net->d_W2));
    CUDA_CHECK(hipFree(net->d_b1));
    CUDA_CHECK(hipFree(net->d_b2));
    free(net);
}

// Main function
int main() {
    printf("MNIST Neural Network - V2 (Optimized Memory Transfers)\n\n");

    // Load datasets
    double** train_images = loadMNISTImages("data/train-images-idx3-ubyte/train-images-idx3-ubyte", 60000);
    double** train_labels = loadMNISTLabels("data/train-labels-idx1-ubyte/train-labels-idx1-ubyte", 60000);
    double** test_images = loadMNISTImages("data/t10k-images-idx3-ubyte/t10k-images-idx3-ubyte", 10000);
    double** test_labels = loadMNISTLabels("data/t10k-labels-idx1-ubyte/t10k-labels-idx1-ubyte", 10000);

    // CPU execution
    NeuralNetwork* net_cpu = createNetwork();
    double cpu_total_time, cpu_loss, cpu_train_acc, cpu_test_acc;
    printf("Running CPU implementation...\n");
    train(net_cpu, train_images, train_labels, 60000, false, &cpu_total_time, &cpu_loss, &cpu_train_acc);
    evaluate(net_cpu, test_images, test_labels, 10000, false, &cpu_test_acc);

    // GPU execution
    NeuralNetwork* net_gpu = createNetwork();
    double gpu_total_time, gpu_loss, gpu_train_acc, gpu_test_acc;
    printf("\nRunning GPU implementation...\n");
    train(net_gpu, train_images, train_labels, 60000, true, &gpu_total_time, &gpu_loss, &gpu_train_acc);
    evaluate(net_gpu, test_images, test_labels, 10000, true, &gpu_test_acc);

    // Result comparison
    printf("\n=== Result Comparison ===\n");
    printf("CPU Total Time: %.3fs\n", cpu_total_time);
    printf("GPU Total Time: %.3fs\n", gpu_total_time);
    printf("Speedup: %.2fx\n", cpu_total_time / gpu_total_time);
    printf("CPU Loss: %.4f\n", cpu_loss);
    printf("GPU Loss: %.4f\n", gpu_loss);
    printf("Loss Difference: %.6f\n", fabs(cpu_loss - gpu_loss));
    printf("CPU Train Accuracy: %.2f%%\n", cpu_train_acc);
    printf("GPU Train Accuracy: %.2f%%\n", gpu_train_acc);
    printf("Train Accuracy Difference: %.2f%%\n", fabs(cpu_train_acc - gpu_train_acc));
    printf("CPU Test Accuracy: %.2f%%\n", cpu_test_acc);
    printf("GPU Test Accuracy: %.2f%%\n", gpu_test_acc);
    printf("Test Accuracy Difference: %.2f%%\n", fabs(cpu_test_acc - gpu_test_acc));

    // Print applied optimizations
    printf("\n=== Applied Memory Optimizations ===\n");
    printf("1. Pinned Host Memory: Used hipHostMalloc for host-side allocations (temp_images, temp_labels, temp_W1, temp_W2) to enable faster host-to-device memory transfers.\n");
    printf("2. Reused Allocated Memory: Allocated d_hidden, d_output, d_d_output, and d_d_hidden once at the start of training and evaluation, reusing them across iterations and epochs, and freeing them only when complete.\n");

    // Free memory
    freeNetwork(net_cpu);
    freeNetwork(net_gpu);
    freeMatrix(train_images, 60000);
    freeMatrix(train_labels, 60000);
    freeMatrix(test_images, 10000);
    freeMatrix(test_labels, 10000);

    return 0;
}