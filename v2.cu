#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.01
#define EPOCHS 3
#define BATCH_SIZE 64
#define NUM_CLASSES 10

// Check CUDA errors
#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Timer function for CPU
double get_cpu_time(clock_t start) {
    return (double)(clock() - start) / CLOCKS_PER_SEC;
}

// Timer function for GPU
float get_gpu_time(hipEvent_t start, hipEvent_t stop) {
    float milliseconds = 0;
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    return milliseconds / 1000.0; // Convert to seconds
}

// Allocate memory for a matrix on host
double** allocateMatrix(int rows, int cols) {
    double** mat = (double**)malloc(rows * sizeof(double*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (double*)malloc(cols * sizeof(double));
    }
    return mat;
}

// Free allocated matrix memory on host
void freeMatrix(double** mat, int rows) {
    for (int i = 0; i < rows; i++) {
        free(mat[i]);
    }
    free(mat);
}

// Activation functions (CPU)
void relu_cpu(double* x, int size) {
    for (int i = 0; i < size; i++) {
        x[i] = (x[i] > 0) ? x[i] : 0;
    }
}

void softmax_cpu(double* x, int size) {
    double max = x[0];
    for (int i = 1; i < size; i++) {
        if (x[i] > max) max = x[i];
    }
    double sum = 0;
    for (int i = 0; i < size; i++) {
        x[i] = exp(x[i] - max);
        sum += x[i];
    }
    for (int i = 0; i < size; i++) {
        x[i] /= sum;
    }
}

// CUDA kernels
__global__ void matrixMulKernel(double* A, double* B, double* C, double* bias, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rowsA && col < colsB) {
        double sum = bias[row];
        for (int k = 0; k < colsA; k++) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

__global__ void reluKernel(double* x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] = (x[idx] > 0) ? x[idx] : 0;
    }
}

__global__ void softmaxKernel(double* x, int size) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        double max = x[0];
        for (int i = 1; i < size; i++) {
            if (x[i] > max) max = x[i];
        }
        double sum = 0;
        for (int i = 0; i < size; i++) {
            x[i] = exp(x[i] - max);
            sum += x[i];
        }
        for (int i = 0; i < size; i++) {
            x[i] /= sum;
        }
    }
}

__global__ void outputGradientKernel(double* output, double* target, double* d_output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_output[idx] = output[idx] - target[idx];
    }
}

__global__ void hiddenGradientKernel(double* W2, double* d_output, double* hidden, double* d_hidden, int hidden_size, int output_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < hidden_size) {
        double sum = 0;
        for (int j = 0; j < output_size; j++) {
            sum += W2[j * hidden_size + i] * d_output[j];
        }
        d_hidden[i] = sum * (hidden[i] > 0);
    }
}

__global__ void updateWeightsKernel(double* W, double* grad, double* input, int rows, int cols, double lr) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows && j < cols) {
        W[i * cols + j] -= lr * grad[i] * input[j];
    }
}

__global__ void updateBiasKernel(double* b, double* grad, int size, double lr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        b[i] -= lr * grad[i];
    }
}

// Neural network structure
typedef struct {
    double** W1;
    double** W2;
    double* b1;
    double* b2;
    double *d_W1, *d_W2, *d_b1, *d_b2;
} NeuralNetwork;

// Initialize neural network
NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    net->W1 = allocateMatrix(HIDDEN_SIZE, INPUT_SIZE);
    net->W2 = allocateMatrix(OUTPUT_SIZE, HIDDEN_SIZE);
    net->b1 = (double*)calloc(HIDDEN_SIZE, sizeof(double));
    net->b2 = (double*)calloc(OUTPUT_SIZE, sizeof(double));

    srand(42);
    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1[i][j] = ((double)rand() / RAND_MAX) * 0.01;

    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2[i][j] = ((double)rand() / RAND_MAX) * 0.01;

    CUDA_CHECK(hipMalloc(&net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->d_b1, HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->d_b2, OUTPUT_SIZE * sizeof(double)));

    double* temp_W1 = (double*)malloc(HIDDEN_SIZE * INPUT_SIZE * sizeof(double));
    double* temp_W2 = (double*)malloc(OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double));
    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            temp_W1[i * INPUT_SIZE + j] = net->W1[i][j];
    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            temp_W2[i * HIDDEN_SIZE + j] = net->W2[i][j];

    CUDA_CHECK(hipMemcpy(net->d_W1, temp_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(net->d_W2, temp_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(net->d_b1, net->b1, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(net->d_b2, net->b2, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

    free(temp_W1);
    free(temp_W2);

    return net;
}

// Forward pass (CPU)
void forward_cpu(NeuralNetwork* net, double* input, double* hidden, double* output) {
    for (int i = 0; i < HIDDEN_SIZE; i++) {
        hidden[i] = net->b1[i];
        for (int j = 0; j < INPUT_SIZE; j++)
            hidden[i] += net->W1[i][j] * input[j];
    }
    relu_cpu(hidden, HIDDEN_SIZE);

    for (int i = 0; i < OUTPUT_SIZE; i++) {
        output[i] = net->b2[i];
        for (int j = 0; j < HIDDEN_SIZE; j++)
            output[i] += net->W2[i][j] * hidden[j];
    }
    softmax_cpu(output, OUTPUT_SIZE);
}

// Forward pass (GPU)
void forward_gpu(NeuralNetwork* net, double* input, double* hidden, double* output) {
    double *d_input, *d_hidden, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, INPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_hidden, HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_output, OUTPUT_SIZE * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_input, input, INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

    dim3 blockDim(16, 16);
    dim3 gridDim((1 + 15) / 16, (HIDDEN_SIZE + 15) / 16);
    matrixMulKernel<<<gridDim, blockDim>>>(net->d_W1, d_input, d_hidden, net->d_b1, HIDDEN_SIZE, INPUT_SIZE, 1);
    CUDA_CHECK(hipDeviceSynchronize());

    int threads = 256;
    int blocks = (HIDDEN_SIZE + threads - 1) / threads;
    reluKernel<<<blocks, threads>>>(d_hidden, HIDDEN_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    gridDim = dim3((1 + 15) / 16, (OUTPUT_SIZE + 15) / 16);
    matrixMulKernel<<<gridDim, blockDim>>>(net->d_W2, d_hidden, d_output, net->d_b2, OUTPUT_SIZE, HIDDEN_SIZE, 1);
    CUDA_CHECK(hipDeviceSynchronize());

    softmaxKernel<<<1, 32>>>(d_output, OUTPUT_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(hidden, d_hidden, HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(output, d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_hidden));
    CUDA_CHECK(hipFree(d_output));
}

// Backward pass (CPU)
void backward_cpu(NeuralNetwork* net, double* input, double* hidden, double* output, double* target) {
    double d_output[OUTPUT_SIZE], d_hidden[HIDDEN_SIZE];

    for (int i = 0; i < OUTPUT_SIZE; i++)
        d_output[i] = output[i] - target[i];

    for (int i = 0; i < HIDDEN_SIZE; i++) {
        d_hidden[i] = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++)
            d_hidden[i] += net->W2[j][i] * d_output[j];
        d_hidden[i] *= (hidden[i] > 0);
    }

    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2[i][j] -= LEARNING_RATE * d_output[i] * hidden[j];

    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1[i][j] -= LEARNING_RATE * d_hidden[i] * input[j];

    for (int i = 0; i < OUTPUT_SIZE; i++)
        net->b2[i] -= LEARNING_RATE * d_output[i];

    for (int i = 0; i < HIDDEN_SIZE; i++)
        net->b1[i] -= LEARNING_RATE * d_hidden[i];
}

// Backward pass (GPU)
void backward_gpu(NeuralNetwork* net, double* input, double* hidden, double* output, double* target) {
    double *d_input, *d_hidden, *d_output, *d_target, *d_d_output, *d_d_hidden;
    CUDA_CHECK(hipMalloc(&d_input, INPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_hidden, HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_output, OUTPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_target, OUTPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_d_output, OUTPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_d_hidden, HIDDEN_SIZE * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_input, input, INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_hidden, hidden, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_output, output, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_target, target, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (OUTPUT_SIZE + threads - 1) / threads;
    outputGradientKernel<<<blocks, threads>>>(d_output, d_target, d_d_output, OUTPUT_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    blocks = (HIDDEN_SIZE + threads - 1) / threads;
    hiddenGradientKernel<<<blocks, threads>>>(net->d_W2, d_d_output, d_hidden, d_d_hidden, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    dim3 blockDim(16, 16);
    dim3 gridDim((HIDDEN_SIZE + 15) / 16, (OUTPUT_SIZE + 15) / 16);
    updateWeightsKernel<<<gridDim, blockDim>>>(net->d_W2, d_d_output, d_hidden, OUTPUT_SIZE, HIDDEN_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());

    gridDim = dim3((INPUT_SIZE + 15) / 16, (HIDDEN_SIZE + 15) / 16);
    updateWeightsKernel<<<gridDim, blockDim>>>(net->d_W1, d_d_hidden, d_input, HIDDEN_SIZE, INPUT_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());

    blocks = (OUTPUT_SIZE + threads - 1) / threads;
    updateBiasKernel<<<blocks, threads>>>(net->d_b2, d_d_output, OUTPUT_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());

    blocks = (HIDDEN_SIZE + threads - 1) / threads;
    updateBiasKernel<<<blocks, threads>>>(net->d_b1, d_d_hidden, HIDDEN_SIZE, LEARNING_RATE);
    CUDA_CHECK(hipDeviceSynchronize());

    double* temp_W1 = (double*)malloc(HIDDEN_SIZE * INPUT_SIZE * sizeof(double));
    double* temp_W2 = (double*)malloc(OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double));
    CUDA_CHECK(hipMemcpy(temp_W1, net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(temp_W2, net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(net->b1, net->d_b1, HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(net->b2, net->d_b2, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1[i][j] = temp_W1[i * INPUT_SIZE + j];
    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2[i][j] = temp_W2[i * HIDDEN_SIZE + j];

    free(temp_W1);
    free(temp_W2);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_hidden));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_target));
    CUDA_CHECK(hipFree(d_d_output));
    CUDA_CHECK(hipFree(d_d_hidden));
}

// Train network
void train(NeuralNetwork* net, double** images, double** labels, int numImages, bool use_gpu, double* total_time, double* loss_out, double* train_acc_out) {
    double loss = 0.0;
    int correct = 0;

    if (use_gpu) {
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        CUDA_CHECK(hipEventRecord(start));

        for (int epoch = 0; epoch < EPOCHS; epoch++) {
            loss = 0.0;
            correct = 0;

            for (int i = 0; i < numImages; i++) {
                double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
                forward_gpu(net, images[i], hidden, output);
                backward_gpu(net, images[i], hidden, output, labels[i]);

                for (int k = 0; k < OUTPUT_SIZE; k++) {
                    loss -= labels[i][k] * log(output[k] + 1e-10);
                }
                int pred = 0, actual = 0;
                for (int j = 0; j < OUTPUT_SIZE; j++) {
                    if (output[j] > output[pred]) pred = j;
                    if (labels[i][j] > labels[i][actual]) actual = j;
                }
                if (pred == actual) correct++;
            }

            printf("GPU Epoch %d - Loss: %.4f - Train Accuracy: %.2f%%\n",
                   epoch + 1, loss / numImages, (correct / (double)numImages) * 100);
            if (epoch == EPOCHS - 1) {
                *loss_out = loss / numImages;
                *train_acc_out = (correct / (double)numImages) * 100;
            }
        }

        CUDA_CHECK(hipEventRecord(stop));
        *total_time = get_gpu_time(start, stop);
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
        printf("GPU Total training time: %.3fs\n", *total_time);
    } else {
        clock_t total_start = clock();

        for (int epoch = 0; epoch < EPOCHS; epoch++) {
            loss = 0.0;
            correct = 0;

            for (int i = 0; i < numImages; i++) {
                double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
                forward_cpu(net, images[i], hidden, output);
                backward_cpu(net, images[i], hidden, output, labels[i]);

                for (int k = 0; k < OUTPUT_SIZE; k++) {
                    loss -= labels[i][k] * log(output[k] + 1e-10);
                }
                int pred = 0, actual = 0;
                for (int j = 0; j < OUTPUT_SIZE; j++) {
                    if (output[j] > output[pred]) pred = j;
                    if (labels[i][j] > labels[i][actual]) actual = j;
                }
                if (pred == actual) correct++;
            }

            printf("CPU Epoch %d - Loss: %.4f - Train Accuracy: %.2f%%\n",
                   epoch + 1, loss / numImages, (correct / (double)numImages) * 100);
            if (epoch == EPOCHS - 1) {
                *loss_out = loss / numImages;
                *train_acc_out = (correct / (double)numImages) * 100;
            }
        }

        *total_time = get_cpu_time(total_start);
        printf("CPU Total training time: %.3fs\n", *total_time);
    }
}

// Evaluate accuracy on test data
void evaluate(NeuralNetwork* net, double** images, double** labels, int numImages, bool use_gpu, double* test_acc_out) {
    int correct = 0;
    for (int i = 0; i < numImages; i++) {
        double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
        if (use_gpu)
            forward_gpu(net, images[i], hidden, output);
        else
            forward_cpu(net, images[i], hidden, output);
        int pred = 0, actual = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            if (output[j] > output[pred]) pred = j;
            if (labels[i][j] > labels[i][actual]) actual = j;
        }
        if (pred == actual) correct++;
    }
    *test_acc_out = (correct / (double)numImages) * 100;
    printf("%s Test Accuracy: %.2f%%\n", use_gpu ? "GPU" : "CPU", *test_acc_out);
}

// Read MNIST dataset
double** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 16, SEEK_SET);
    double** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;
            if (fread(&pixel, sizeof(unsigned char), 1, file) != 1) {
                fprintf(stderr, "Error: Failed to read pixel\n");
                fclose(file);
                exit(EXIT_FAILURE);
            }
            images[i][j] = pixel / 255.0;
        }
    }
    fclose(file);
    return images;
}

double** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 8, SEEK_SET);
    double** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        if (fread(&label, sizeof(unsigned char), 1, file) != 1) {
            fprintf(stderr, "Error: Failed to read label\n");
            fclose(file);
            exit(EXIT_FAILURE);
        }
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0 : 0.0;
        }
    }
    fclose(file);
    return labels;
}

// Free network memory
void freeNetwork(NeuralNetwork* net) {
    freeMatrix(net->W1, HIDDEN_SIZE);
    freeMatrix(net->W2, OUTPUT_SIZE);
    free(net->b1);
    free(net->b2);
    CUDA_CHECK(hipFree(net->d_W1));
    CUDA_CHECK(hipFree(net->d_W2));
    CUDA_CHECK(hipFree(net->d_b1));
    CUDA_CHECK(hipFree(net->d_b2));
    free(net);
}

// Main function
int main() {
    printf("MNIST Neural Network - V2 (Naive GPU with Fixed Timing)\n\n");

    // Load datasets
    double** train_images = loadMNISTImages("/home/bscs-22i-1210/snap/snapd-desktop-integration/current/Desktop/project-root_HPC/data/train-images-idx3-ubyte/train-images-idx3-ubyte", 60000);
    double** train_labels = loadMNISTLabels("/home/bscs-22i-1210/snap/snapd-desktop-integration/current/Desktop/project-root_HPC/data/train-labels-idx1-ubyte/train-labels-idx1-ubyte", 60000);
    double** test_images = loadMNISTImages("/home/bscs-22i-1210/snap/snapd-desktop-integration/current/Desktop/project-root_HPC/data/t10k-images-idx3-ubyte/t10k-images-idx3-ubyte", 10000);
    double** test_labels = loadMNISTLabels("/home/bscs-22i-1210/snap/snapd-desktop-integration/current/Desktop/project-root_HPC/data/t10k-labels-idx1-ubyte/t10k-labels-idx1-ubyte", 10000);

    // CPU execution
    NeuralNetwork* net_cpu = createNetwork();
    double cpu_total_time, cpu_loss, cpu_train_acc, cpu_test_acc;
    printf("Running CPU implementation...\n");
    train(net_cpu, train_images, train_labels, 60000, false, &cpu_total_time, &cpu_loss, &cpu_train_acc);
    evaluate(net_cpu, test_images, test_labels, 10000, false, &cpu_test_acc);

    // GPU execution
    NeuralNetwork* net_gpu = createNetwork();
    double gpu_total_time, gpu_loss, gpu_train_acc, gpu_test_acc;
    printf("\nRunning GPU implementation...\n");
    train(net_gpu, train_images, train_labels, 60000, true, &gpu_total_time, &gpu_loss, &gpu_train_acc);
    evaluate(net_gpu, test_images, test_labels, 10000, true, &gpu_test_acc);

    // Result comparison
    printf("\n=== Result Comparison ===\n");
    printf("CPU Total Time: %.3fs\n", cpu_total_time);
    printf("GPU Total Time: %.3fs\n", gpu_total_time);
    printf("Speedup: %.2fx\n", cpu_total_time / gpu_total_time);
    printf("CPU Loss: %.4f\n", cpu_loss);
    printf("GPU Loss: %.4f\n", gpu_loss);
    printf("Loss Difference: %.6f\n", fabs(cpu_loss - gpu_loss));
    printf("CPU Train Accuracy: %.2f%%\n", cpu_train_acc);
    printf("GPU Train Accuracy: %.2f%%\n", gpu_train_acc);
    printf("Train Accuracy Difference: %.2f%%\n", fabs(cpu_train_acc - gpu_train_acc));
    printf("CPU Test Accuracy: %.2f%%\n", cpu_test_acc);
    printf("GPU Test Accuracy: %.2f%%\n", gpu_test_acc);
    printf("Test Accuracy Difference: %.2f%%\n", fabs(cpu_test_acc - gpu_test_acc));

    // Free memory
    freeNetwork(net_cpu);
    freeNetwork(net_gpu);
    freeMatrix(train_images, 60000);
    freeMatrix(train_labels, 60000);
    freeMatrix(test_images, 10000);
    freeMatrix(test_labels, 10000);

    return 0;
}